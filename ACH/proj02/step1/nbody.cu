#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xstehl14
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

#define BLOCK_SIZE 128

__device__ void particles_pos_dev(t_particles p_out, float dt, int i);

__global__ void particles_simulate(t_particles p_in, t_particles p_out, int N, float dt, const float GDT)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float4 pos_i = p_in.pos[i];
	float3 vel_i = p_in.vel[i];
	float3 d;

	for (int tile = 0; tile < gridDim.x; tile++) {
		__shared__ float4 spos_j[BLOCK_SIZE];
		spos_j[threadIdx.x] = p_in.pos[tile * blockDim.x + threadIdx.x];
		__syncthreads();      

		#pragma unroll 4
		for (int j = 0; j < BLOCK_SIZE; j++) {
			// Calculate distance between two points in each axis
			d.x = spos_j[j].x - pos_i.x;
			d.y = spos_j[j].y - pos_i.y;
			d.z = spos_j[j].z - pos_i.z;

			// Calculate vector distance between two points
			float dist_R = rsqrtf(d.x * d.x +  d.y*d.y + d.z*d.z + FLT_EPSILON);
			//float inv_dist = rsqrtf(dist_R);
			float inv_dist_3R = dist_R * dist_R * dist_R;

			float F = GDT * spos_j[j].w * inv_dist_3R;

			//vel_i.x += F * d.x;
			vel_i.x = fmaf(F, d.x, vel_i.x);
			//vel_i.y += F * d.y;
			vel_i.y = fmaf(F, d.y, vel_i.y);
			//vel_i.z += F + d.z;
			vel_i.z = fmaf(F, d.z, vel_i.z);
		}
		__syncthreads();
	}


	p_out.vel[i] = vel_i;
	p_out.pos[i].x = fmaf(p_out.vel[i].x,dt, p_in.pos[i].x);
	p_out.pos[i].y = fmaf(p_out.vel[i].y,dt, p_in.pos[i].y);
	p_out.pos[i].z = fmaf(p_out.vel[i].z,dt, p_in.pos[i].z);
}

__global__ void particles_pos(t_particles p_in, t_particles p_out, float dt)
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
			__syncthreads();
}

void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fscanf(fp, "%f %f %f %f %f %f %f \n",
			&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
            &p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
            &p.pos[i].w);
    }
}

void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
            p.pos[i].x, p.pos[i].y, p.pos[i].z,
            p.vel[i].x, p.vel[i].y, p.vel[i].z,
            p.pos[i].w);
    }
}
