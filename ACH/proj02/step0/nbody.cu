#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xlogin00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void particles_simulate(t_particles p_in, t_particles p_out, int N, float dt)
{
    // ZDE DOPLNTE TELO KERNELU
}

void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fscanf(fp, "%f %f %f %f %f %f %f \n",
            // ZDE DOPLNTE NACTENI JEDNE CASTICE
    }
}

void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
            // ZDE DOPLNTE VYPSANI JEDNE CASTICE
    }
}
