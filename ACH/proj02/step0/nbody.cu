#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xlogin00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__device__ void particles_pos_dev(t_particles p_out, float dt, int i);

__global__ void particles_simulate(t_particles p_in, t_particles p_out, int N, float dt, const float GDT)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float3 pos_i = p_in.pos[i];
	float4 vel_i = p_in.vel[i];

	//#pragma unroll 128
	for (int j = 0; j < N; j++) {
		// Calculate distance between two points in each axis
		float3 d;
			
		d.x = p_in.pos[j].x - pos_i.x;
		d.y = p_in.pos[j].y - pos_i.y;
		d.z = p_in.pos[j].z - pos_i.z;

		// Calculate vector distance between two points
		float dist_R = d.x * d.x +  d.y*d.y + d.z*d.z + FLT_EPSILON;
		float inv_dist = rsqrtf(dist_R);
		float inv_dist_3R = inv_dist * inv_dist * inv_dist;

		float F = GDT * p_in.vel[j].w * inv_dist_3R;

		vel_i.x = fmaf(F, d.x, vel_i.x);
		vel_i.y = fmaf(F, d.y, vel_i.y);
		vel_i.z = fmaf(F, d.z, vel_i.z);
	}

	__syncthreads();

	p_out.vel[i] = vel_i;
	p_out.pos[i].x = fmaf(p_out.vel[i].x,dt, p_in.pos[i].x);
	p_out.pos[i].y = fmaf(p_out.vel[i].y,dt, p_in.pos[i].y);
	p_out.pos[i].z = fmaf(p_out.vel[i].z,dt, p_in.pos[i].z);


}

__global__ void particles_pos(t_particles p_in, t_particles p_out, float dt)
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
			__syncthreads();
}

void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fscanf(fp, "%f %f %f %f %f %f %f \n",
			&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
            &p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
            &p.vel[i].w);
    }
}

void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
            p.pos[i].x, p.pos[i].y, p.pos[i].z,
            p.vel[i].x, p.vel[i].y, p.vel[i].z,
            p.vel[i].w);
    }
}
