#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xstehl14
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <iostream>
#include <algorithm>

#include "nbody.h"

using namespace std;

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

	int blocksPerGrid = (thr_blc + N -1) / thr_blc;

	if (blocksPerGrid == 0) {
		blocksPerGrid = 1;
		thr_blc = N;
	}

	if (thr_blc > N) {
		thr_blc = N;
	}
	cout << "blocks: " << blocksPerGrid << endl; 
	cout << "threads/block: " << thr_blc << endl; 

	//const size_t size = N * sizeof(float);
    // alokace pameti na CPU
    t_particles particles_cpu;
    int size = N/thr_blc;
    if (N % thr_blc != 0)
    	size++;

    size *= thr_blc;

    cout << "size of array: " << size << endl;

    /* Host allocation */
    hipHostAlloc(&particles_cpu.pos, size * sizeof(float3),hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, size * sizeof(float4), hipHostMallocDefault);
	
	const float GDT = G * dt;

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[2]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu[2];
    for (int i = 0; i < 2; i++)
    {
        // alokace pameti na GPU
        // ZDE DOPLNTE ALOKACI PAMETI NA GPU

		hipMalloc(&(particles_gpu[i].pos), size * sizeof(float3));
		hipMalloc(&(particles_gpu[i].vel), size * sizeof(float4));

        // kopirovani castic na GPU
        // ZDE DOPLNTE KOPIROVANI DAT Z CPU NA GPU
		hipMemcpy(particles_gpu[i].pos, particles_cpu.pos,
				size * sizeof(float3), hipMemcpyHostToDevice);
		hipMemcpy(particles_gpu[i].vel, particles_cpu.vel,
				size * sizeof(float4), hipMemcpyHostToDevice);
    }

    // vypocet
    gettimeofday(&t1, 0);
    for (int s = 0; s < steps; s++)
    {
        // ZDE DOPLNTE SPUSTENI KERNELU
		particles_simulate <<<blocksPerGrid, thr_blc>>> (particles_gpu[0], particles_gpu[1], N, dt, GDT);
		//particles_pos<<<blocksPerGrid, thr_blc>>> (particles_gpu[0], particles_gpu[1], dt);
		t_particles tmp = particles_gpu[0];
		particles_gpu[1] = particles_gpu[0];
		particles_gpu[0] = tmp;
    }
    // ZDE DOPLNTE SYNCHRONIZACI
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

	// check for error
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    // kpirovani castic zpet na CPU
    // ZDE DOPLNTE KOPIROVANI DAT Z GPU NA CPU
	hipMemcpy(particles_cpu.pos, particles_gpu[1].pos,
			N * sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.vel, particles_gpu[1].vel,
			N * sizeof(float4), hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

	for (int i = 0; i < 2; i++) {
		hipFree(&particles_gpu[i]);
	}

	hipHostFree(&particles_cpu);
	hipFree(&particles_gpu[0]);
	hipFree(&particles_gpu[1]);

    return 0;
}
