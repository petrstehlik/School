/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xstehl14
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <iostream>
#include <algorithm>

#include "nbody.h"

using namespace std;

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

	const size_t size = N * sizeof(float);
    // alokace pameti na CPU
    t_particles particles_cpu;

    /* Host allocation */
    hipHostAlloc(&particles_cpu.pos_x, size, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.pos_y, size, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.pos_z, size, hipHostMallocDefault);

    hipHostAlloc(&particles_cpu.vel_x, size, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel_y, size, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel_z, size, hipHostMallocDefault);

    hipHostAlloc(&particles_cpu.weight, size, hipHostMallocDefault);

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[2]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu[2];
    for (int i = 0; i < 2; i++)
    {
        // alokace pameti na GPU
        // ZDE DOPLNTE ALOKACI PAMETI NA GPU

		hipMalloc(&(particles_gpu[i].pos_x), size);
		hipMalloc(&(particles_gpu[i].pos_y), size);
		hipMalloc(&(particles_gpu[i].pos_z), size);

		hipMalloc(&(particles_gpu[i].vel_x), size);
		hipMalloc(&(particles_gpu[i].vel_y), size);
		hipMalloc(&(particles_gpu[i].vel_z), size);

		hipMalloc(&(particles_gpu[i].weight), size);

        // kopirovani castic na GPU
        // ZDE DOPLNTE KOPIROVANI DAT Z CPU NA GPU
		hipMemcpy(particles_gpu[i].pos_x, particles_cpu.pos_x, size, hipMemcpyHostToDevice);
		hipMemcpy(particles_gpu[i].pos_y, particles_cpu.pos_y, size, hipMemcpyHostToDevice);
		hipMemcpy(particles_gpu[i].pos_z, particles_cpu.pos_z, size, hipMemcpyHostToDevice);

		hipMemcpy(particles_gpu[i].vel_x, particles_cpu.vel_x, size, hipMemcpyHostToDevice);
		hipMemcpy(particles_gpu[i].vel_y, particles_cpu.vel_y, size, hipMemcpyHostToDevice);
		hipMemcpy(particles_gpu[i].vel_z, particles_cpu.vel_z, size, hipMemcpyHostToDevice);
		
		hipMemcpy(particles_gpu[i].weight, particles_cpu.weight, size, hipMemcpyHostToDevice);
    }

	int blocksPerGrid = (thr_blc + N -1) / thr_blc;

	cout << "blocks: " << blocksPerGrid << endl; 
	if (blocksPerGrid == 0) {
		blocksPerGrid = 1;
		thr_blc = N;
	}

	if (thr_blc > N) {
		thr_blc = N;
	}

	cout << "blocks: " << blocksPerGrid << endl; 
	cout << "threads: " << thr_blc << endl; 

	//blocksPerGrid = 256;

	const float GDT = G * dt;

    // vypocet
    gettimeofday(&t1, 0);
    for (int s = 0; s < steps; s++)
    {
        // ZDE DOPLNTE SPUSTENI KERNELU
		particles_simulate <<<blocksPerGrid, thr_blc>>> (particles_gpu[1], particles_gpu[1], N, dt, GDT);
		particles_pos<<<blocksPerGrid, thr_blc>>> (particles_gpu[1], dt);

		//particles_simulate <<<1, N>>> (particles_gpu[1], particles_gpu[1], N, dt, GDT);
		//particles_pos<<<1, N>>> (particles_gpu[1], dt);
		//std::swap(particles_gpu[0], particles_gpu[1]);

    }
    // ZDE DOPLNTE SYNCHRONIZACI
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

	// check for error
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    // kpirovani castic zpet na CPU
    // ZDE DOPLNTE KOPIROVANI DAT Z GPU NA CPU
	hipMemcpy(particles_cpu.pos_x, particles_gpu[1].pos_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.pos_y, particles_gpu[1].pos_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.pos_z, particles_gpu[1].pos_z, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.vel_x, particles_gpu[1].vel_x, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.vel_y, particles_gpu[1].vel_y, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.vel_z, particles_gpu[1].vel_z, size, hipMemcpyDeviceToHost);
	hipMemcpy(particles_cpu.weight, particles_gpu[1].weight, size, hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

	for (int i = 0; i < 2; i++) {
		hipFree(&particles_gpu[i]);
	}

	hipHostFree(&particles_cpu);
	hipFree(&particles_gpu[0]);
	hipFree(&particles_gpu[1]);

    return 0;
}
