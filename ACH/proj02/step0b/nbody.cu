#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xlogin00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void particles_simulate(t_particles p_in, t_particles p_out, int N, float dt, const float GDT)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float pos_x_i = p_in.pos_x[i];
	float pos_y_i = p_in.pos_y[i];
	float pos_z_i = p_in.pos_z[i];
	float vel_x_i = p_in.vel_x[i];
	float vel_y_i = p_in.vel_y[i];
	float vel_z_i = p_in.vel_z[i];

	#pragma unroll
	for (int j = 0; j < N; j++) {
		if (i == j)
			continue;
		// Calculate distance between two points in each axis
		float dx = p_in.pos_x[j] - pos_x_i;
		float dy = p_in.pos_y[j] - pos_y_i;
		float dz = p_in.pos_z[j] - pos_z_i;

		// Calculate vector distance between two points
		float dist_R = dx*dx + dy*dy + dz*dz;
		//float dist_R = norm3df(dx,dy,dz);
		float inv_dist = rsqrtf(dist_R);
		float inv_dist_3R = inv_dist * inv_dist * inv_dist;

		float F = (GDT * p_in.weight[j]) * inv_dist_3R;//powf(R, 3);
		//float s = GDT * p_in.weight[j] * inv_dist;

		//vel_x_i = fmaf(F, dx, vel_x_i);
		//vel_y_i = fmaf(F, dx, vel_x_i);
		//vel_z_i = fmaf(F, dx, vel_x_i);

		vel_x_i += F * dx;
		vel_y_i += F * dy;
		vel_z_i += F * dz;

	}

	//__syncthreads();

	p_out.vel_x[i] = vel_x_i;
	p_out.vel_y[i] = vel_y_i;
	p_out.vel_z[i] = vel_z_i;

	//p_out.pos_x[i] += p_out.vel_x[i] * dt;
	//p_out.pos_y[i] += p_out.vel_y[i] * dt;
	//p_out.pos_z[i] += p_out.vel_z[i] * dt;

}

__global__ void particles_pos(t_particles p_out, float dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	p_out.pos_x[i] += p_out.vel_x[i] * dt;
	p_out.pos_y[i] += p_out.vel_y[i] * dt;
	p_out.pos_z[i] += p_out.vel_z[i] * dt;
}

void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fscanf(fp, "%f %f %f %f %f %f %f \n",
            &p.pos_x[i], &p.pos_y[i], &p.pos_z[i],
            &p.vel_x[i], &p.vel_y[i], &p.vel_z[i],
            &p.weight[i]);
    }
}

void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
            p.pos_x[i], p.pos_y[i], p.pos_z[i],
            p.vel_x[i], p.vel_y[i], p.vel_z[i],
            p.weight[i]);
    }
}
