#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2016)
 * Projekt c. 2 (cuda)
 * Login: xstehl14
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

#define BLOCK_SIZE 128

__device__ void particles_pos_dev(t_particles p_out, float dt, int i);

__global__ void particles_simulate(t_particles p_in, t_particles p_out, const int N, const float dt, const float GDT)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float4 pos_i = p_in.pos[i];
	float3 d;

	float3 F_i = {0.0f, 0.0f, 0.0f};

	for (int tile = 0; tile < gridDim.x; tile++) {
		__shared__ float4 spos_j[BLOCK_SIZE];
		spos_j[threadIdx.x] = p_in.pos[tile * blockDim.x + threadIdx.x];
		__syncthreads();      

		#pragma unroll 4
		for (int j = 0; j < BLOCK_SIZE; j++) {
			// Calculate distance between two points in each axis
			d.x = spos_j[j].x - pos_i.x;
			d.y = spos_j[j].y - pos_i.y;
			d.z = spos_j[j].z - pos_i.z;

			// Calculate vector distance between two points
			float dist_R = rsqrtf(d.x * d.x +  d.y*d.y + d.z*d.z + FLT_EPSILON);

			float F = GDT * spos_j[j].w * dist_R * dist_R * dist_R;

			F_i.x = fmaf(F, d.x, F_i.x);
			F_i.y = fmaf(F, d.y, F_i.y);
			F_i.z = fmaf(F, d.z, F_i.z);
		}
		__syncthreads();
	}

	p_out.vel[i].x = p_in.vel[i].x + F_i.x;
	p_out.vel[i].y = p_in.vel[i].y + F_i.y;
	p_out.vel[i].z = p_in.vel[i].z + F_i.z;

	p_out.pos[i].x = fmaf(p_out.vel[i].x, dt, p_in.pos[i].x);
	p_out.pos[i].y = fmaf(p_out.vel[i].y, dt, p_in.pos[i].y);
	p_out.pos[i].z = fmaf(p_out.vel[i].z, dt, p_in.pos[i].z);
}

void particles_read(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fscanf(fp, "%f %f %f %f %f %f %f \n",
			&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
            &p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
            &p.pos[i].w);
    }
}

void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
            p.pos[i].x, p.pos[i].y, p.pos[i].z,
            p.vel[i].x, p.vel[i].y, p.vel[i].z,
            p.pos[i].w);
    }
}
